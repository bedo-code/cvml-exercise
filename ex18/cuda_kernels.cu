#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"

__global__ void convertToFloat(const unsigned char* input, float* output, 
                             int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        output[y * width + x] = static_cast<float>(input[y * width + x]);
    }
}

__global__ void horizontalScan(const float* input, float* output, 
                             int width, int height) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (y < height) {
        // Use shared memory for efficient scanning
        extern __shared__ float temp[];
        
        // Load data into shared memory
        for (int x = threadIdx.x; x < width; x += blockDim.x) {
            temp[x] = input[y * width + x];
        }
        __syncthreads();
        
        // Perform exclusive scan in shared memory
        for (int offset = 1; offset < width; offset *= 2) {
            float t;
            if (threadIdx.x >= offset) {
                t = temp[threadIdx.x - offset];
            }
            __syncthreads();
            if (threadIdx.x >= offset) {
                temp[threadIdx.x] += t;
            }
            __syncthreads();
        }
        
        // Write results to global memory
        for (int x = threadIdx.x; x < width; x += blockDim.x) {
            output[y * width + x] = temp[x];
        }
    }
}

__global__ void verticalScan(const float* input, float* output, 
                            int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (x < width) {
        extern __shared__ float temp[];
        
        // Load data into shared memory
        for (int y = threadIdx.y; y < height; y += blockDim.y) {
            temp[y] = input[y * width + x];
        }
        __syncthreads();
        
        // Perform exclusive scan in shared memory
        for (int offset = 1; offset < height; offset *= 2) {
            float t;
            if (threadIdx.y >= offset) {
                t = temp[threadIdx.y - offset];
            }
            __syncthreads();
            if (threadIdx.y >= offset) {
                temp[threadIdx.y] += t;
            }
            __syncthreads();
        }
        
        // Write results to global memory
        for (int y = threadIdx.y; y < height; y += blockDim.y) {
            output[y * width + x] = temp[y];
        }
    }
}